#include "hip/hip_runtime.h"
/******************************************************************************
*                                                                             *
* Distributed Hash Cracker v2.0                                               *
*                                                                             *
* Copyright (c) 2009 RPISEC.                                                  *
* All rights reserved.                                                        *
*                                                                             *
* Redistribution and use in source and binary forms, with or without modifi-  *
* cation, are permitted provided that the following conditions are met:       *
*                                                                             *
*    * Redistributions of source code must retain the above copyright notice  *
*      this list of conditions and the following disclaimer.                  *
*                                                                             *
*    * Redistributions in binary form must reproduce the above copyright      *
*      notice, this list of conditions and the following disclaimer in the    *
*      documentation and/or other materials provided with the distribution.   *
*                                                                             *
*    * Neither the name of RPISEC nor the names of its contributors may be    *
*      used to endorse or promote products derived from this software without *
*      specific prior written permission.                                     *
*                                                                             *
* THIS SOFTWARE IS PROVIDED BY RPISEC "AS IS" AND ANY EXPRESS OR IMPLIED      *
* WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF        *
* MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN     *
* NO EVENT SHALL RPISEC BE HELD LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,  *
* SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED    *
* TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR      *
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF      *
* LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING        *
* NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS          *
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.                *
*                                                                             *
*******************************************************************************
*                                                                             *
* CrackThread.cu - main code goes here                                        *
*                                                                             *
******************************************************************************/

#include "CrackThread-CUDA.h"
#include "../Cracker-common/HashingAlgorithm.h"
#include "../Cracker-common/BaseNInteger.h"
#include "../ComputeNode/CrackThread.h"

#include "config.h"

using namespace std;

double GetTime();

__global__ void sanityKernel1(int* bOK)
{
	*bOK = 1;	
}

__global__ void sanityKernel2(int* bOK)
{
	*bOK = 2;	
}

typedef void (*CUDA_HASHPROC)(int*, unsigned char*, unsigned char*, int, unsigned char*, int*, int, int,int);

#if WINDOWS
BOOL APIENTRY DllMain( HMODULE hModule,
                       DWORD  ul_reason_for_call,
                       LPVOID lpReserved
					 )
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
	case DLL_THREAD_ATTACH:
	case DLL_THREAD_DETACH:
	case DLL_PROCESS_DETACH:
		break;
	}
	return TRUE;
}
#endif

int GetOptimalThreadCount(HashingAlgorithm* pAlg);

THREAD_PROTOTYPE(internalComputeThreadProc,_pData)
{
	try
	{
		//Cache some settings
		CrackThreadData* pData = reinterpret_cast<CrackThreadData*>(_pData);
		BaseNInteger start=pData->start;
		BaseNInteger end=pData->end;
		int guesslen = start.GetSize();
		string target=pData->target;
		volatile bool& bDoneWithWU = *pData->bDoneWithWU;
		
		//Convert target from hex to binary
		HashingAlgorithm* pAlg = pData->pAlg;
		int hashsize = pAlg->GetHashLength();
		unsigned char* btarget = new unsigned char[hashsize];
		for(unsigned int i=0;i<hashsize;i++)
		{
			int b;
			sscanf(target.substr(i*2,2).c_str(),"%x",&b);
			btarget[i]=b;
		}
		
		//Set our CUDA device
		hipError_t err = hipSuccess;
		if(hipSuccess != (err = hipSetDevice(pData->tid)) )
			throw string("Failed to set CUDA device: ") + hipGetErrorString(err);
			
		//Run a quick test to make sure that the CUDA driver is set up properly for this device
		int* d_sanitybuf;
		if(hipSuccess != (err = hipMalloc(reinterpret_cast<void**>(&d_sanitybuf), 4)) )
			throw string("Failed to allocate GPU memory for sanity check: ") + hipGetErrorString(err);
		int bSanityCheck = 0xCC;
		sanityKernel1<<<128,128>>>(d_sanitybuf);
		if(hipSuccess != (err = hipMemcpy(&bSanityCheck, d_sanitybuf, 4, hipMemcpyDeviceToHost)) )
			throw string("Failed to copy sanity check") + hipGetErrorString(err);
		if(bSanityCheck != 1)
		{
			char err[1024];
			sprintf(
				err,
				"CUDA sanity check failed: read 0x%x, expected 0x1. Please verify that you have "
				"a CUDA-compatible graphics card and install the latest CUDA drivers from "
				"http://www.nvidia.com/object/cuda_get.html."
				,bSanityCheck);
			throw string(err);
		}
		sanityKernel2<<<128,128>>>(d_sanitybuf);
		if(hipSuccess != (err = hipMemcpy(&bSanityCheck, d_sanitybuf, 4, hipMemcpyDeviceToHost)) )
			throw string("Failed to copy sanity check") + hipGetErrorString(err);
		if(bSanityCheck != 2)
		{
			char err[1024];
			sprintf(
				err,
				"CUDA sanity check failed: read 0x%x, expected 0x2. Please verify that you have "
				"a CUDA-compatible graphics card and install the latest CUDA drivers from "
				"http://www.nvidia.com/object/cuda_get.html."
				,bSanityCheck);
			throw string(err);
		}
		hipFree(d_sanitybuf);
		d_sanitybuf=NULL;
			
		//Get pointer to hashing algorithm
		CUDA_HASHPROC DoHash = reinterpret_cast<CUDA_HASHPROC>(pAlg->Hash);

		//Get optimal thread count (will be set to 0 if not known, in which case we need to calculate it)
		int threads = GetOptimalThreadCount(pAlg);
		int blocks = 32768;
			
 		//Copy charset to GPU
 		unsigned char* d_charset=NULL;
		if(hipSuccess != (err = hipMalloc(reinterpret_cast<void**>(&d_charset), pData->charset.length())) )
			throw string("Failed to allocate GPU memory for charset: ") + hipGetErrorString(err);
		if(hipSuccess != (err = hipMemcpy(d_charset, pData->charset.c_str(), pData->charset.length(), hipMemcpyHostToDevice)) )
				throw string("Failed to copy charset buffer: ") + hipGetErrorString(err);
		
		//Allocate GPU memory for start guess
		int* d_startguess=NULL;
		if(hipSuccess != (err = hipMalloc(reinterpret_cast<void**>(&d_startguess), guesslen*sizeof(int))) )
			throw string("Failed to allocate GPU memory for start guess: ") + hipGetErrorString(err);

		//Allocate GPU memory for cracked value
		unsigned char* d_cracked = NULL;
		int* d_bCracked = NULL;
		if(hipSuccess != (err = hipMalloc(reinterpret_cast<void**>(&d_cracked), 32)) )
			throw string("Failed to allocate GPU memory for cracked value: ") + hipGetErrorString(err);
		if(hipSuccess != (err = hipMalloc(reinterpret_cast<void**>(&d_bCracked), 4)) )
			throw string("Failed to allocate GPU memory for crack flag: ") + hipGetErrorString(err);
		int zero = 0;
		if(hipSuccess != (err = hipMemcpy(d_bCracked, &zero, 4, hipMemcpyHostToDevice)) )
			throw string("Failed to copy zero: ") + hipGetErrorString(err);
			
		//Allocate and copy target
		unsigned char* d_target = NULL;
		if(hipSuccess != (err = hipMalloc(reinterpret_cast<void**>(&d_target), hashsize)) )
			throw string("Failed to allocate GPU memory for target: ") + hipGetErrorString(err);
		if(hipSuccess != (err = hipMemcpy(d_target, btarget, hashsize, hipMemcpyHostToDevice)) )
			throw string("Failed to copy target: ") + hipGetErrorString(err);
		
		int* startdigits = &start.digits[0];
		
		//Don't have optimal settings stored? Run a little test
		if(threads==0)
		{
			cout << endl;
			cout << "Algorithm " << pAlg->GetName() << " does not have optimal settings in config file, performing benchmark" << endl;
		
			int optThreads = 1;
			float optSpeed = 0;

			//TODO: allow use of more threads on devices with >8192 regs
			int testBlocks = 4096;
			for(threads=16; threads <= 255; threads++)
			{
				//Copy start guess to GPU memory
				double start = GetTime();
				if(hipSuccess != (err = hipMemcpy(d_startguess, startdigits, guesslen * sizeof(int), hipMemcpyHostToDevice)) )
					throw string("Failed to copy start-guess buffer: ") + hipGetErrorString(err);
		
				//Test this thread count
				DoHash(
					d_startguess,
					d_target,
					d_charset,
					(int)pData->charset.length(),
					d_cracked,
					d_bCracked,
					guesslen,
					testBlocks,
					threads);
				
				//Copy some data back so we wait for the kernel
				int bCracked = 0;
				if(hipSuccess != (err = hipMemcpy(&bCracked, d_bCracked, 4, hipMemcpyDeviceToHost)) )
					throw string("Failed to copy cracked flag: ") + hipGetErrorString(err);
					
				double dt = GetTime()-start;
				float speed = static_cast<float>(threads*testBlocks)/(1E6 * dt);
				
				//See what turns out best
				if(speed > optSpeed)
				{
					optSpeed = speed;
					optThreads = threads;
				}
				
				if((threads & 0xF) == 0)
				{
					cout << ".";
					fflush(stdout);
				}
			}
			
			FILE* fp = fopen("cudathreads.conf","a");
			fprintf(fp,"%s %d\n",pAlg->GetName(),optThreads);
			fclose(fp);
			
			cout << endl << "Optimal thread count is " << optThreads << " (speed = " << optSpeed << " MHz)" << endl;
			threads = optThreads;
		}
		
		//Avoid overkill for short WUs
		int totalHashes = static_cast<int>(end.toInt() - start.toInt());
		if(threads*blocks > totalHashes)
			blocks = ceil(static_cast<float>(totalHashes) / threads);
			
		//Loop until we're done with the work unit
		while(!bDoneWithWU)
		{
			/*
			cout << "Starting kernel with " << blocks << " blocks of " << threads << " threads... " << endl;
			cout << "\t";
			for(int i=0;i<guesslen; i++)
				cout << start.digits[i] << " ";
			cout << endl;
			*/
		
			//Copy start guess to GPU memory
			double tStart=GetTime();
			if(hipSuccess != (err = hipMemcpy(d_startguess, startdigits, guesslen * sizeof(int), hipMemcpyHostToDevice)) )
				throw string("Failed to copy start-guess buffer: ") + hipGetErrorString(err);
			if(hipSuccess != (err = hipMemcpy(d_bCracked, &zero, sizeof(int), hipMemcpyHostToDevice)) )
				throw string("Failed to copy bCracked: ") + hipGetErrorString(err);
		
			//Process it
			DoHash(
				d_startguess,
				d_target,
				d_charset,
				(int)pData->charset.length(),
				d_cracked,
				d_bCracked,
				guesslen,
				blocks,
				threads);
		
			//Bump our index while waiting for GPU
			start.AddWithSaturation(blocks*threads);
			
			//See what came of it
			int bCracked = 0;
			if(hipSuccess != (err = hipMemcpy(&bCracked, d_bCracked, sizeof(int), hipMemcpyDeviceToHost)) )
				throw string("Failed to copy cracked flag: ") + hipGetErrorString(err);
			//printf("\tbCracked = %d\n",bCracked);
			if(bCracked)
			{
				//We found it! Copy it back from the device.
				char* sCracked = new char[guesslen];
				if(hipSuccess != (err = hipMemcpy(sCracked, d_cracked, guesslen, hipMemcpyDeviceToHost)) )
					throw string("Failed to copy cracked value: ") + hipGetErrorString(err);
					
				//Save the string and quit the crack loop
				pData->crackval.clear();
				for(int q=0; q<guesslen; q++)
					pData->crackval += static_cast<char>(sCracked[q]);
				pData->bFound=true;
				bDoneWithWU=true;
				delete[] sCracked;
				break;
			}
			
			/*
			double tEnd = GetTime();
			double dt=tEnd-tStart;
			float speed = static_cast<float>(threads*blocks)/(1E6 * dt);
			cout << "\tdone (in " << dt << " seconds, " << speed << " MHz)" << endl;
			fflush(stdout);
			*/
			
			//Done with last guess? Quit
			if(! (start < end) )
				break;
		}

		//Clean up
		delete[] btarget;
		hipFree(d_cracked);
		hipFree(d_bCracked);
		hipFree(d_cracked);
		hipFree(d_charset);
		hipFree(d_startguess);
	}
	catch(std::string err)
	{
		cerr << "ERROR: " << err.c_str() << endl;
		exit(1);
	}

	//Done
	THREAD_RETURN(0);
}

bool Initialize()
{
	//See how many devices we have
	int count;
	if(hipSuccess != hipGetDeviceCount(&count))
	{
		cout << "Failed to get CUDA device count" << endl;
		return false;
	}

	//Print some info, hiding settings we don't care about for now
	hipDeviceProp_t prop;
	for(int i=0;i<count;i++)
	{
		hipGetDeviceProperties(&prop,i);
		cout << "Device " << i << " : " << prop.name << endl;
		/*cout << "\tGlobal mem (MB):       " << prop.totalGlobalMem / (1024*1024) << endl;
		//cout << "\tShared mem per block:  " << prop.sharedMemPerBlock << endl;
		cout << "\tRegs per block:        " << prop.regsPerBlock << endl;
		//cout << "\tWarp size:             " << prop.warpSize << endl;
		//cout << "\tMem pitch:             " << prop.memPitch << endl;
		cout << "\tMax threads per block: " << prop.maxThreadsPerBlock << endl;
		//cout << "\tConst mem:             " << prop.totalConstMem << endl;
		cout << "\tCompute capability:    " << prop.major << "." << prop.minor << endl;
		cout << "\tClock rate (MHz):      " << (float)prop.clockRate / 1000 << endl;
		cout << "\tMultiprocessor count:  " << prop.multiProcessorCount << endl;
		*/
	}
 
	return true;
}

void Cleanup()
{
	//No global cleanup - CUDA resources will be automatically freed at termination
}

int GetCrackType()
{
	return CRACK_TYPE_GPU;
}

int GetMaxThreads()
{
	//One thread per GPU
	int count;
	if(hipSuccess != hipGetDeviceCount(&count))
	{
		cout << "Failed to get CUDA device count" << endl;
		exit(-1);
	}
	return count;
}

ZTHREADPROC GetComputeThread()
{
	//Get our compute thread
	return internalComputeThreadProc;
}

int GetOptimalThreadCount(HashingAlgorithm* pAlg)
{
	//Do we have it already? Take a look;
	ifstream fin("cudathreads.conf");
	if(!fin)
		return 0;
	while(!fin.eof())
	{
		char name[128];
		int threads;
		fin >> name;
		fin >> threads;
		if(string(name) == pAlg->GetName())
			return threads;
	}
	return 0;
}

double GetTime()
{
#if LINUX
	timespec t;
	clock_gettime(CLOCK_REALTIME,&t);
	double d = static_cast<double>(t.tv_nsec) / 1E9f;
	d += t.tv_sec;
	return d;

#elif BSD
	timeval t;
	gettimeofday(&t, NULL);
	double d = static_cast<double>(t.tv_usec) / 1E6f;
	d += t.tv_sec;
	return d;

#elif WINDOWS
	static __int64 freq;
	static bool init=false;
	if(!init)
	{
		init=true;
		QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
	}

	__int64 t;
	QueryPerformanceCounter((LARGE_INTEGER*)&t);
	return static_cast<double>(t)/freq;
#else
#error Unrecognized platform. Please define either LINUX, BSD, or WINDOWS.
#endif
}
